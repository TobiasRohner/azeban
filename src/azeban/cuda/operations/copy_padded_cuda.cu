#include "hip/hip_runtime.h"
/*
 * This file is part of azeban (https://github.com/TobiasRohner/azeban).
 * Copyright (c) 2021 Tobias Rohner.
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */
#include <azeban/config.hpp>
#include <azeban/operations/copy_padded.hpp>
#include <zisa/config.hpp>

namespace azeban {

__global__ void
copy_to_padded_cuda_kernel(zisa::array_view<complex_t, 1> dst,
                           zisa::array_const_view<complex_t, 1> src,
                           complex_t pad_value) {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < src.shape(0)) {
    dst[idx] = src[idx];
  } else if (idx < dst.shape(0)) {
    dst[idx] = pad_value;
  }
}

__global__ void
copy_to_padded_cuda_kernel(zisa::array_view<complex_t, 2> dst,
                           zisa::array_const_view<complex_t, 2> src,
                           complex_t pad_value) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  const auto src_shape = src.shape();
  const auto dst_shape = dst.shape();
  const int idx_dst = zisa::row_major<2>::linear_index(dst_shape, i, j);
  if (j < src_shape[1]) {
    if (i < dst_shape[0]) {
      if (i < src_shape[0] / 2 + 1) {
        const int idx_src = zisa::row_major<2>::linear_index(src_shape, i, j);
        dst[idx_dst] = src[idx_src];
      } else if (i < src_shape[0] / 2 + 1 + dst_shape[0] - src_shape[0]) {
        dst[idx_dst] = pad_value;
      } else {
        const int idx_src = zisa::row_major<2>::linear_index(
            src_shape, i + src_shape[0] - dst_shape[0], j);
        dst[idx_dst] = src[idx_src];
      }
    }
  } else if (i < dst_shape[0] && j < dst_shape[1]) {
    dst[idx_dst] = pad_value;
  }
}

__global__ void
copy_to_padded_cuda_kernel(zisa::array_view<complex_t, 3> dst,
                           zisa::array_const_view<complex_t, 3> src,
                           complex_t pad_value) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  const int k = blockIdx.z * blockDim.z + threadIdx.z;
  const auto src_shape = src.shape();
  const auto dst_shape = dst.shape();
  const int idx_dst = zisa::row_major<3>::linear_index(dst_shape, i, j, k);
  int i_src, j_src;

  if (i >= dst_shape[0] || j >= dst_shape[1] || k >= dst_shape[2]) {
    return;
  }

  if (k >= src_shape[2]) {
    dst[idx_dst] = pad_value;
    return;
  }

  if (j < src_shape[1] / 2 + 1) {
    j_src = j;
  } else if (j < src_shape[1] / 2 + 1 + dst_shape[1] - src_shape[1]) {
    dst[idx_dst] = pad_value;
    return;
  } else {
    j_src = j + src_shape[1] - dst_shape[1];
  }

  if (i < src_shape[0] / 2 + 1) {
    i_src = i;
  } else if (i < src_shape[0] / 2 + 1 + dst_shape[0] - src_shape[0]) {
    dst[idx_dst] = pad_value;
    return;
  } else {
    i_src = i + src_shape[0] - dst_shape[0];
  }

  const int idx_src
      = zisa::row_major<3>::linear_index(src_shape, i_src, j_src, k);
  dst[idx_dst] = src[idx_src];
}

void copy_to_padded_cuda(const zisa::array_view<complex_t, 1> &dst,
                         const zisa::array_const_view<complex_t, 1> &src,
                         const complex_t &pad_value) {
  assert(src.memory_location() == zisa::device_type::cuda);
  assert(dst.memory_location() == zisa::device_type::cuda);
  assert(dst.shape(0) >= src.shape(0));
  const int thread_dims = 1024;
  const int block_dims = zisa::min(
      zisa::div_up(static_cast<int>(dst.shape(0)), thread_dims), 1024);
  copy_to_padded_cuda_kernel<<<block_dims, thread_dims>>>(dst, src, pad_value);
  ZISA_CHECK_CUDA_DEBUG;
  hipDeviceSynchronize();
}

void copy_to_padded_cuda(const zisa::array_view<complex_t, 2> &dst,
                         const zisa::array_const_view<complex_t, 2> &src,
                         const complex_t &pad_value) {
  assert(src.memory_location() == zisa::device_type::cuda);
  assert(dst.memory_location() == zisa::device_type::cuda);
  assert(dst.shape(0) >= src.shape(0));
  assert(dst.shape(1) >= src.shape(1));
  const dim3 thread_dims(32, 32, 1);
  const dim3 block_dims(
      zisa::min(zisa::div_up(static_cast<int>(dst.shape(0)), thread_dims.x),
                1024),
      zisa::min(zisa::div_up(static_cast<int>(dst.shape(1)), thread_dims.y),
                1024),
      1);
  copy_to_padded_cuda_kernel<<<block_dims, thread_dims>>>(dst, src, pad_value);
  ZISA_CHECK_CUDA_DEBUG;
  hipDeviceSynchronize();
}

void copy_to_padded_cuda(const zisa::array_view<complex_t, 3> &dst,
                         const zisa::array_const_view<complex_t, 3> &src,
                         const complex_t &pad_value) {
  assert(src.memory_location() == zisa::device_type::cuda);
  assert(dst.memory_location() == zisa::device_type::cuda);
  assert(dst.shape(0) >= src.shape(0));
  assert(dst.shape(1) >= src.shape(1));
  assert(dst.shape(2) >= src.shape(2));
  const dim3 thread_dims(4, 4, 32);
  const dim3 block_dims(
      zisa::min(zisa::div_up(static_cast<int>(dst.shape(0)), thread_dims.x),
                1024),
      zisa::min(zisa::div_up(static_cast<int>(dst.shape(1)), thread_dims.y),
                1024),
      zisa::min(zisa::div_up(static_cast<int>(dst.shape(2)), thread_dims.z),
                1024));
  copy_to_padded_cuda_kernel<<<block_dims, thread_dims>>>(dst, src, pad_value);
  ZISA_CHECK_CUDA_DEBUG;
  hipDeviceSynchronize();
}

}
